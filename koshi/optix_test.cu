#include "hip/hip_runtime.h"
#include <optix.h>
#include <optix_device.h>

#include <koshi/RenderOptix.h>
#include <koshi/IntersectList.h>

using namespace Koshi;

extern "C" 
{
__constant__ Koshi::Resources resources;
}

DEVICE_FUNCTION IntersectList * unpackIntersects()
{ 
    const uint32_t ptr0 = optixGetPayload_0();
    const uint32_t ptr1 = optixGetPayload_1();
    const uint64_t ptr = static_cast<uint64_t>(ptr0) << 32 | ptr1;
    return reinterpret_cast<IntersectList*>(ptr); 
}

extern "C" __global__ void __raygen__rg() 
{
    // Lookup our location within the launch grid
    const uint3 idx = optixGetLaunchIndex();
    const uint3 dim = optixGetLaunchDimensions();

    const Ray ray = resources.camera->sample(idx.x, idx.y);
    IntersectList intersects = resources.intersector->intersect(ray);

    if(intersects.size() > 0)
    {
        Intersect& intersect = intersects[0];
        resources.aovs[0].write(Vec2u(idx.x, idx.y), Vec4f(intersect.uvw0[0], intersect.uvw0[1], 0.f, 1.f));
    }
}

extern "C" __global__ void __miss__ms() 
{
}

extern "C" __global__ void __closesthit__ch() 
{
    IntersectList * intersects = unpackIntersects();
    Intersect& intersect = intersects->push();
    float2 uvs = optixGetTriangleBarycentrics();
    intersect.uvw0 = Vec3f(uvs.x, uvs.y, 0.f);
}